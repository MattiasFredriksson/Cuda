﻿#include "CudaFuncs.h"
#include <cuda_gl_interop.h>
#include <hip/device_functions.h>
#include ""
#include<iostream>
#include <assert.h>

#pragma region Functions

__device__ int div2ceil(int value) { return (value & 1) + (value >> 1); }
__device__ int cuda_div_ceil(int nume, int denom) { return nume / denom + ((nume % denom) > 0); }
/* For positive nums with sum less then INT_MAX*/
__device__ int cuda_div_ceil_pos(int nume, int denom) { return (nume + denom - 1) / denom; }

#pragma endregion



#pragma region Device constructs

/* Initiate runtime device. */
bool initCudaDevice()
{
	hipError_t err = hipSetDevice(0);
	if (err != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return false;
	}
	return true;
}
__host__ void cudaCheck()
{
	hipDeviceSynchronize();
	hipError_t err = hipPeekAtLastError();
	if (err != hipSuccess)
	{
		std::cout << "Error: " << hipGetErrorString(err) << std::endl;
		hipDeviceReset();
		exit(0);
	}
}

bool read(int* dev_arr, int* arr, size_t arr_len)
{
	// Copy output vector from GPU buffer to host memory.
	hipError_t cudaStatus = hipMemcpy(arr, dev_arr, arr_len * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return false;
	}
	return true;
}
bool read(float* dev_arr, float* arr, size_t arr_len)
{
	// Copy output vector from GPU buffer to host memory.
	hipError_t cudaStatus = hipMemcpy(arr, dev_arr, arr_len * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return false;
	}
	return true;
}


#pragma endregion

#ifdef OPEN_GL

bool allocateTexture_RGBA(unsigned int width, unsigned int height, CU_image &image)
{
	// Generate a texture ID
	glGenTextures(1, &image._textureID);
	// Make this the current texture (remember that GL is state-based)
	glBindTexture(GL_TEXTURE_2D, image._textureID);
	// Allocate the texture memory. The last parameter is NULL since we only
	// want to allocate memory, not initialize it
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, width, height, 0, GL_RGBA,
		GL_FLOAT, NULL);
	// Must set the filter mode, GL_LINEAR enables interpolation when scaling
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);

	GLenum gl_err = glGetError();
	hipError_t err = hipGraphicsGLRegisterImage(&image._resource, image._textureID, GL_TEXTURE_2D, 
		hipGraphicsRegisterFlagsSurfaceLoadStore);
	if (err != hipSuccess)
	{
		glDeleteBuffers(1, &image._textureID);
		fprintf(stderr, "hipGraphicsGLRegisterImage failed!\n");
		return false;
	}

	return true;
}

hipError_t createCudaSurface(hipArray_t arr, hipSurfaceObject_t &surfObj)
{
	//Create resource desc.
	struct hipResourceDesc resDesc; 
	memset(&resDesc, 0, sizeof(resDesc)); 
	resDesc.resType = hipResourceTypeArray; 
	resDesc.res.array.array = arr;
	
	return hipCreateSurfaceObject(&surfObj, &resDesc);
}

hipError_t CU_image::map(hipArray_t &arr)
{
	hipError_t err = hipGraphicsMapResources(1, &_resource);
	if (err != hipSuccess)
		return err;
	return hipGraphicsSubResourceGetMappedArray(&arr, _resource, 0, 0);
}


hipError_t CU_image::mapSurface(hipSurfaceObject_t &surfObj)
{
	hipArray_t arr;
	hipError_t err = map(arr);
	if (err != hipSuccess) return err;
	err = createCudaSurface(arr, surfObj);
	return err;
}
hipError_t CU_image::unmap()
{
	return hipGraphicsUnmapResources(1, &_resource);
}
hipError_t CU_image::destroy()
{
	hipError_t err = hipGraphicsUnregisterResource(_resource);
	glDeleteBuffers(1, &_textureID);
	_textureID = 0;
	return err;
}



#endif